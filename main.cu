#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

#include "poseidon.cuh"

#define N 10

void hostPoseidon(F* states) {
    F state[WIDTH];

    for (int i=0; i<N; i++) {
        for (int j=0; j<WIDTH; j++) {
            state[j] = states[i*WIDTH + j];
        }

        poseidon(state);

        for (int j=0; j<WIDTH; j++) {
            states[i*WIDTH + j] = state[j];
        }
    }

    return;
}

__global__ void devicePoseidon(F* states) {
    F state[WIDTH];

    int i = threadIdx.x;
    for (int j=0; j<WIDTH; j++) {
        state[j] = states[i*WIDTH + j];
    }

    poseidon(state);

    for (int j=0; j<WIDTH; j++) {
        states[i*WIDTH + j] = state[j];
    }

    return;
}

void print_debug(F* states) {
    std::cout << std::hex;
    for (int i=0; i<N; i++) {
        for (int j=0; j<WIDTH; j++) {
            std::cout << states[i*WIDTH + j] << ", ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

int main() {
    F* states = (F*)malloc(sizeof(F)*N*WIDTH);

    /******
       Init
    ******/
    for (int i=0; i<N; i++) {
        for (int j=0; j<WIDTH; j++) {
            states[i*WIDTH + j] = F(0);
        }
    }
    print_debug(states);

    /******
       Host
    ******/
    hostPoseidon(states);
    print_debug(states);

    /******
       Init
    ******/
    for (int i=0; i<N; i++) {
        for (int j=0; j<WIDTH; j++) {
            states[i*WIDTH + j] = F(0);
        }
    }
    print_debug(states);

    /********
       Device
    ********/
    F* d_states;
    hipMalloc(&d_states, sizeof(F)*N*WIDTH);
    hipMemcpy(d_states, states, sizeof(F)*N*WIDTH, hipMemcpyHostToDevice);

    devicePoseidon<<<1, N>>>(d_states);

    F* returned_states = (F*)malloc(sizeof(F)*N*WIDTH);
    hipMemcpy(returned_states, d_states, sizeof(F)*N*WIDTH, hipMemcpyDeviceToHost);
    print_debug(returned_states);

    hipFree(d_states);

    return 0;
}
