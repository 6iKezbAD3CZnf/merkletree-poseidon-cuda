#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>

#include "merkle_tree.cuh"

__host__ __device__
void two_to_one(F* digest, F* left, F* right) {
    F state[SPONGE_WIDTH] = { F(0) };

    for (int k=0; k<SPONGE_WIDTH; k++) {
        if (k < HASH_WIDTH) {
            // left
            state[k] = left[k];
        } else if (k < 2*HASH_WIDTH) {
            // right
            state[k] = right[k - HASH_WIDTH];
        } else {
            state[k] = F(0);
        }
    }

    poseidon(state);

    for (int k=0; k<HASH_WIDTH; k++) {
        digest[k] = state[k];
    }
}

// __host__ __device__
// void permute(F* digest, F* leave) {
//     F state[SPONGE_WIDTH] = { F(0) };

//     for (int k=0; k<SPONGE_WIDTH; k++) {
//         if (k < LEAVE_WIDTH) {
//             state[k] = leave[k];
//         } else {
//             break;
//         }
//     }

//     poseidon(state);

//     for (int k=0; k<HASH_WIDTH; k++) {
//         digest[k] = state[k];
//     }
// }

__host__ __device__
void hash_or_noop(F* digest, F* leave, uint32_t leave_len) {
    if (leave_len * 8 <= HASH_WIDTH) {
        assert(false);
    }

    // hash_no_pad()
    // hash_n_to_hash_no_pad()
    // hash_n_to_m_no_pad()
    F state[SPONGE_WIDTH] = { F(0) };

    uint32_t quo = leave_len / SPONGE_RATE;
    uint32_t rem = leave_len % SPONGE_RATE;
    for (uint32_t i=0; i<quo; i++) {
        for (uint32_t j=0; j<SPONGE_RATE; j++) {
            state[j] = leave[i*SPONGE_RATE + j];
        }
        poseidon(state);
    }
    if (rem) {
        for (uint32_t i=0; i<rem; i++) {
            state[i] = leave[quo*SPONGE_RATE + i];
        }
        poseidon(state);
    }

    for (uint32_t i=0; i<HASH_WIDTH; i++) {
        digest[i] = state[i];
    }
}

__global__
void device_fill_digests0(F* d_digests, F* d_leaves, uint32_t n_leaves, uint32_t leave_len) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (i < n_leaves) {
        uint32_t from = i;
        uint32_t to = (i>>1<<2) | (i&0b1);
        hash_or_noop(d_digests + to*HASH_WIDTH, d_leaves + from*leave_len, leave_len);
        i += stride;
    }
}

__global__
void device_fill_digests1(
        F* d_digests,
        uint32_t level,
        uint32_t n_level_leaves,
        uint32_t last_level_start_idx,
        uint32_t level_start_idx
) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (i < n_level_leaves) {
        uint32_t left = last_level_start_idx + i*(1<<(level+1));
        uint32_t right = left + 1;
        uint32_t to = (level_start_idx + (i>>1)*(1<<(level+2))) | (i&0b1);
        two_to_one(d_digests + to*HASH_WIDTH, d_digests + left*HASH_WIDTH, d_digests + right*HASH_WIDTH);

        i += stride;
    }

    return;
}

void device_fill_digests_cap(
        F* digests_cap,
        uint32_t n_digests_cap,
        F* leaves,
        uint32_t n_leaves,
        uint32_t leave_len,
        uint32_t cap_height
        ) {
    F* d_digests;
    F* d_leaves;
    hipMalloc(&d_leaves, sizeof(F)*leave_len*n_digests_cap);
    hipMalloc(&d_digests, sizeof(F)*HASH_WIDTH*n_digests_cap);
    hipMemcpy(d_leaves, leaves, sizeof(F)*leave_len*n_leaves, hipMemcpyHostToDevice);

    device_fill_digests0<<<N_BLOCK, N_THREAD>>>(d_digests, d_leaves, n_leaves, leave_len);
    hipDeviceSynchronize();

    int level = 1;
    int n_level_leaves = n_leaves >> 1;
    uint32_t last_level_start_idx = 0;
    uint32_t level_start_idx = 2;

    while (n_level_leaves > (1 << cap_height)) {
        device_fill_digests1<<<N_BLOCK, N_THREAD>>>(d_digests, level, n_level_leaves, last_level_start_idx, level_start_idx);
        hipDeviceSynchronize();

        level += 1;
        n_level_leaves = n_level_leaves >> 1;
        last_level_start_idx = level_start_idx;
        level_start_idx += (1<<level);
    }

    hipMemcpy(digests_cap, d_digests, sizeof(F)*HASH_WIDTH*n_digests_cap, hipMemcpyDeviceToHost);

    hipFree(d_leaves);
    hipFree(d_digests);

    // caps
    uint32_t left = last_level_start_idx;
    uint32_t right = left + 1;
    uint32_t to = level_start_idx;
    two_to_one(digests_cap + to*HASH_WIDTH, digests_cap + left*HASH_WIDTH, digests_cap + right*HASH_WIDTH);

    return;
}

void host_fill_digests_cap(
        F* digests_cap,
        F* leaves,
        uint32_t n_leaves,
        uint32_t leave_len,
        uint32_t cap_height
        ) {
    F state[SPONGE_WIDTH] = { F(0) };

    for (uint32_t i=0; i<n_leaves; i++) {
        uint32_t from = i;
        uint32_t to = (i>>1<<2) | (i&0b1);
        hash_or_noop(digests_cap + to*HASH_WIDTH, leaves + from*leave_len, leave_len);
    }

    uint32_t level = 1;
    uint32_t n_level_leaves = n_leaves >> 1;
    uint32_t last_level_start_idx = 0;
    uint32_t level_start_idx = 2;

    while (n_level_leaves > (uint32_t) (1 << cap_height)) {
        for (uint32_t i=0; i<n_level_leaves; i++) {
            uint32_t left = last_level_start_idx + i*(1<<(level+1));
            uint32_t right = left + 1;
            uint32_t to = (level_start_idx + (i>>1)*(1<<(level+2))) | (i&0b1);
            two_to_one(digests_cap + to*HASH_WIDTH, digests_cap + left*HASH_WIDTH, digests_cap + right*HASH_WIDTH);
        }

        level += 1;
        n_level_leaves = n_level_leaves >> 1;
        last_level_start_idx = level_start_idx;
        level_start_idx += (1<<level);
    }

    // caps
    uint32_t left = last_level_start_idx;
    uint32_t right = left + 1;
    uint32_t to = level_start_idx;
    two_to_one(digests_cap + to*HASH_WIDTH, digests_cap + left*HASH_WIDTH, digests_cap + right*HASH_WIDTH);

    return;
}

void print_leaves(F* leaves, uint32_t n_leaves, uint32_t leave_len) {
    for (uint32_t i=0; i<n_leaves; i++) {
        std::cout << std::dec;
        std::cout << "leave" << i << " is [";
        std::cout << std::hex;
        for (uint32_t j=0; j<leave_len; j++) {
            std::cout << leaves[i*leave_len + j] << ", ";
        }
        std::cout << "]" << std::endl;
    }
    std::cout << std::endl;
    std::cout << std::dec;
}

void print_digests(F* digests, uint32_t n_digests) {
    for (uint32_t i=0; i<n_digests; i++) {
        std::cout << std::dec;
        std::cout << "digest" << i << " is [";
        std::cout << std::hex;
        for (int j=0; j<HASH_WIDTH; j++) {
            std::cout << digests[i*HASH_WIDTH + j] << ", ";
        }
        std::cout << "]" << std::endl;
    }
    std::cout << std::endl;
    std::cout << std::dec;
}

void print_cap(F* digests_cap, uint32_t n_digests, uint32_t cap_height) {
    std::cout << std::hex;
    for (int i=0; i<(1<<cap_height); i++) {
        for (int j=0; j<HASH_WIDTH; j++) {
            std::cout << digests_cap[(n_digests+i)*HASH_WIDTH + j] << ", ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
    std::cout << std::dec;
}
