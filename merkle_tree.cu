#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>

#include "merkle_tree.cuh"

__host__ __device__
void two_to_one(F* digest, F* left, F* right) {
    F state[SPONGE_WIDTH] = { F(0) };

    for (int k=0; k<SPONGE_WIDTH; k++) {
        if (k < HASH_WIDTH) {
            // left
            state[k] = left[k];
        } else if (k < 2*HASH_WIDTH) {
            // right
            state[k] = right[k - HASH_WIDTH];
        } else {
            state[k] = F(0);
        }
    }

    poseidon(state);

    for (int k=0; k<HASH_WIDTH; k++) {
        digest[k] = state[k];
    }
}

__host__ __device__
void hash_or_noop(F* digest, F* leave, uint32_t leave_len) {
    if (leave_len <= HASH_WIDTH) {
        // noop
        for (uint32_t i=0; i<HASH_WIDTH; i++) {
            if (i < leave_len) {
                digest[i] = leave[i];
            } else {
                digest[i] = F(0);
            }
        }

        return;
    }

    // hash_no_pad()
    // hash_n_to_hash_no_pad()
    // hash_n_to_m_no_pad()
    F state[SPONGE_WIDTH] = { F(0) };

    uint32_t quo = leave_len / SPONGE_RATE;
    uint32_t rem = leave_len % SPONGE_RATE;
    for (uint32_t i=0; i<quo; i++) {
        for (uint32_t j=0; j<SPONGE_RATE; j++) {
            state[j] = leave[i*SPONGE_RATE + j];
        }
        poseidon(state);
    }
    if (rem) {
        for (uint32_t i=0; i<rem; i++) {
            state[i] = leave[quo*SPONGE_RATE + i];
        }
        poseidon(state);
    }

    for (uint32_t i=0; i<HASH_WIDTH; i++) {
        digest[i] = state[i];
    }
}

__global__
void device_fill_digests0(
        F* d_digests_caps,
        uint32_t num_subtree_digests,
        F* d_leaves,
        uint32_t num_subtree_leaves,
        uint32_t leave_len,
        uint32_t num_caps
        ) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (id < num_caps * num_subtree_leaves) {
        int j = id % num_subtree_leaves; // outer loop
        int i = (id - j) / num_subtree_leaves; // inner loop

        uint32_t from = j;
        uint32_t to = (j>>1<<2) | (j&0b1);
        hash_or_noop(
                d_digests_caps + (num_subtree_digests*i + to)*HASH_WIDTH,
                d_leaves + (num_subtree_leaves*i + from)*leave_len,
                leave_len
                );

        id += stride;
    }
}

__global__
void device_fill_digests1(
        F* d_digests_caps,
        uint32_t num_subtree_digests,
        uint32_t level,
        uint32_t num_level_subtree_digests,
        uint32_t last_level_start_idx,
        uint32_t level_start_idx,
        uint32_t num_caps
) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (id < num_caps * num_level_subtree_digests) {
        int j = id % num_level_subtree_digests; // outer loop
        int i = (id - j) / num_level_subtree_digests; // inner loop

        uint32_t left = last_level_start_idx + j*(1<<(level+1));
        uint32_t right = left + 1;
        uint32_t to = (level_start_idx + (j>>1)*(1<<(level+2))) | (j&0b1);
        two_to_one(
                d_digests_caps + (num_subtree_digests*i + to)*HASH_WIDTH,
                d_digests_caps + (num_subtree_digests*i + left)*HASH_WIDTH,
                d_digests_caps + (num_subtree_digests*i + right)*HASH_WIDTH
                );

        id += stride;
    }

    return;
}

void device_fill_digests_caps(
        F* digests_caps,
        uint32_t num_digests,
        F* leaves,
        uint32_t num_leaves,
        uint32_t leave_len,
        uint32_t cap_height
        ) {
    uint32_t num_caps = 1 << cap_height;

    F* d_digests_caps;
    F* d_leaves;
    hipMalloc(&d_leaves, sizeof(F)*leave_len*num_leaves);
    hipMalloc(&d_digests_caps, sizeof(F)*HASH_WIDTH*(num_digests + num_caps));
    hipMemcpy(d_leaves, leaves, sizeof(F)*leave_len*num_leaves, hipMemcpyHostToDevice);

    device_fill_digests0<<<N_BLOCK, N_THREAD>>>(
            d_digests_caps,
            num_digests / num_caps,
            d_leaves,
            num_leaves / num_caps,
            leave_len,
            num_caps
            );
    hipDeviceSynchronize();

    int level = 1;
    uint32_t num_level_digests = num_leaves >> 1;
    uint32_t last_level_start_idx = 0;
    uint32_t level_start_idx = 2;
    while (num_level_digests > num_caps) {
        device_fill_digests1<<<N_BLOCK, N_THREAD>>>(
                d_digests_caps,
                num_digests / num_caps,
                level,
                num_level_digests / num_caps,
                last_level_start_idx,
                level_start_idx,
                num_caps
                );
        hipDeviceSynchronize();

        level += 1;
        num_level_digests = num_level_digests >> 1;
        last_level_start_idx = level_start_idx;
        level_start_idx += (1<<level);
    }

    hipMemcpy(digests_caps, d_digests_caps, sizeof(F)*HASH_WIDTH*(num_digests + num_caps), hipMemcpyDeviceToHost);

    hipFree(d_leaves);
    hipFree(d_digests_caps);

    // caps
    for (uint32_t i=0; i<num_caps; i++) {
        uint32_t subtree_digests_idx = num_digests / num_caps * i;
        uint32_t left = last_level_start_idx;
        uint32_t right = left + 1;
        two_to_one(
                digests_caps + (num_digests + i)*HASH_WIDTH,
                digests_caps + (subtree_digests_idx + left)*HASH_WIDTH,
                digests_caps + (subtree_digests_idx + right)*HASH_WIDTH
                );
    }

    return;
}

void host_fill_digests_caps_sub(
        uint32_t subtree_leaves_idx,
        uint32_t subtree_digests_idx,
        uint32_t cap_idx,
        F* digests_caps,
        F* leaves,
        uint32_t num_leaves,
        uint32_t leave_len
        ) {
    for (uint32_t i=0; i<num_leaves; i++) {
        uint32_t from = i;
        uint32_t to = (i>>1<<2) | (i&0b1);
        hash_or_noop(
                digests_caps + (subtree_digests_idx + to)*HASH_WIDTH,
                leaves + (subtree_leaves_idx + from)*leave_len,
                leave_len
                );
    }

    uint32_t level = 1;
    uint32_t num_level_leaves = num_leaves >> 1;
    uint32_t last_level_start_idx = 0;
    uint32_t level_start_idx = 2;

    while (num_level_leaves > 1) {
        for (uint32_t i=0; i<num_level_leaves; i++) {
            uint32_t left = last_level_start_idx + i*(1<<(level+1));
            uint32_t right = left + 1;
            uint32_t to = (level_start_idx + (i>>1)*(1<<(level+2))) | (i&0b1);
            two_to_one(
                    digests_caps + (subtree_digests_idx + to)*HASH_WIDTH,
                    digests_caps + (subtree_digests_idx + left)*HASH_WIDTH,
                    digests_caps + (subtree_digests_idx + right)*HASH_WIDTH
                    );
        }

        level += 1;
        num_level_leaves = num_level_leaves >> 1;
        last_level_start_idx = level_start_idx;
        level_start_idx += (1<<level);
    }

    // caps
    uint32_t left = last_level_start_idx;
    uint32_t right = left + 1;
    two_to_one(
            digests_caps + cap_idx*HASH_WIDTH,
            digests_caps + (subtree_digests_idx + left)*HASH_WIDTH,
            digests_caps + (subtree_digests_idx + right)*HASH_WIDTH
            );

    return;
}

void host_fill_digests_caps(
        F* digests_caps,
        uint32_t num_digests,
        F* leaves,
        uint32_t num_leaves,
        uint32_t leave_len,
        uint32_t cap_height
) {
    uint32_t num_caps = 1 << cap_height;
    uint32_t num_subtree_leaves = num_leaves / num_caps;
    uint32_t num_subtree_digests = num_digests / num_caps;
    for (uint32_t i=0; i<num_caps; i++) {
        host_fill_digests_caps_sub(
            num_subtree_leaves * i,
            num_subtree_digests * i,
            num_digests + i,
            digests_caps,
            leaves,
            num_leaves / num_caps,
            leave_len
        );
    }

    return;
}

void print_leaves(F* leaves, uint32_t num_leaves, uint32_t leave_len) {
    for (uint32_t i=0; i<num_leaves; i++) {
        std::cout << std::dec;
        std::cout << "leave" << i << " is [";
        std::cout << std::hex;
        for (uint32_t j=0; j<leave_len; j++) {
            std::cout << leaves[i*leave_len + j] << ", ";
        }
        std::cout << "]" << std::endl;
    }
    std::cout << std::endl;
    std::cout << std::dec;
}

void print_digests(F* digests, uint32_t num_digests) {
    for (uint32_t i=0; i<num_digests; i++) {
        std::cout << std::dec;
        std::cout << "digest" << i << " is [";
        std::cout << std::hex;
        for (int j=0; j<HASH_WIDTH; j++) {
            std::cout << digests[i*HASH_WIDTH + j] << ", ";
        }
        std::cout << "]" << std::endl;
    }
    std::cout << std::endl;
    std::cout << std::dec;
}

void print_caps(F* digests_caps, uint32_t num_digests, uint32_t cap_height) {
    std::cout << std::hex;
    for (int i=0; i<(1<<cap_height); i++) {
        for (int j=0; j<HASH_WIDTH; j++) {
            std::cout << digests_caps[(num_digests+i)*HASH_WIDTH + j] << ", ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
    std::cout << std::dec;
}
