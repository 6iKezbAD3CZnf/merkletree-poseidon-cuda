#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>

#include "merkle_tree.cuh"

__host__ __device__
void two_to_one(F* digests, uint32_t left, uint32_t right, uint32_t to) {
    F state[SPONGE_WIDTH] = { F(0) };

    for (int k=0; k<SPONGE_WIDTH; k++) {
        if (k < HASH_WIDTH) {
            // left
            state[k] = digests[left*HASH_WIDTH + k];
        } else if (k < 2*HASH_WIDTH) {
            // right
            state[k] = digests[right*HASH_WIDTH + k - HASH_WIDTH];
        } else {
            state[k] = F(0);
        }
    }

    poseidon(state);

    for (int k=0; k<HASH_WIDTH; k++) {
        digests[to*HASH_WIDTH + k] = state[k];
    }
}

__host__ __device__
void permute(F* digests, F* leaves, uint32_t from, uint32_t to) {
    F state[SPONGE_WIDTH];

    for (int k=0; k<SPONGE_WIDTH; k++) {
        if (k < LEAVE_WIDTH) {
            state[k] = leaves[from*LEAVE_WIDTH + k];
        } else {
            state[k] = F(0);
        }
    }

    poseidon(state);

    for (int k=0; k<HASH_WIDTH; k++) {
        digests[to*HASH_WIDTH + k] = state[k];
    }
}

__global__
void device_fill_digests0(F* d_digests, F* d_leaves, uint32_t n_leaves) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (i < n_leaves) {
        uint32_t from = i;
        uint32_t to = (i>>1<<2) | (i&0b1);
        permute(d_digests, d_leaves, from, to);
        i += stride;
    }
}

__global__
void device_fill_digests1(
        F* d_digests,
        uint32_t level,
        uint32_t n_level_leaves,
        uint32_t last_level_start_idx,
        uint32_t level_start_idx
) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (i < n_level_leaves) {
        uint32_t left = last_level_start_idx + i*(1<<(level+1));
        uint32_t right = left + 1;
        uint32_t to = (level_start_idx + (i>>1)*(1<<(level+2))) | (i&0b1);
        two_to_one(d_digests, left, right, to);

        i += stride;
    }

    return;
}

void device_fill_digests_cap(F* digests_cap, uint32_t n_digests_cap, F* leaves, uint32_t n_leaves, uint32_t cap_height) {
    F* d_digests;
    F* d_leaves;
    hipMalloc(&d_leaves, sizeof(F)*LEAVE_WIDTH*n_digests_cap);
    hipMalloc(&d_digests, sizeof(F)*HASH_WIDTH*n_digests_cap);
    hipMemcpy(d_leaves, leaves, sizeof(F)*LEAVE_WIDTH*n_leaves, hipMemcpyHostToDevice);

    device_fill_digests0<<<N_BLOCK, N_THREAD>>>(d_digests, d_leaves, n_leaves);
    hipDeviceSynchronize();

    int level = 1;
    int n_level_leaves = n_leaves >> 1;
    uint32_t last_level_start_idx = 0;
    uint32_t level_start_idx = 2;

    while (n_level_leaves > (1 << cap_height)) {
        device_fill_digests1<<<N_BLOCK, N_THREAD>>>(d_digests, level, n_level_leaves, last_level_start_idx, level_start_idx);
        hipDeviceSynchronize();

        level += 1;
        n_level_leaves = n_level_leaves >> 1;
        last_level_start_idx = level_start_idx;
        level_start_idx += (1<<level);
    }

    hipMemcpy(digests_cap, d_digests, sizeof(F)*HASH_WIDTH*n_digests_cap, hipMemcpyDeviceToHost);

    hipFree(d_leaves);
    hipFree(d_digests);

    // caps
    uint32_t left = last_level_start_idx;
    uint32_t right = left + 1;
    uint32_t to = level_start_idx;
    two_to_one(digests_cap, left, right, to);

    return;
}

void host_fill_digests_cap(F* digests_cap, F* leaves, uint32_t n_leaves, uint32_t cap_height) {
    F state[SPONGE_WIDTH] = { F(0) };

    for (uint32_t i=0; i<n_leaves; i++) {
        uint32_t from = i;
        uint32_t to = (i>>1<<2) | (i&0b1);
        permute(digests_cap, leaves, from, to);
    }

    uint32_t level = 1;
    uint32_t n_level_leaves = n_leaves >> 1;
    uint32_t last_level_start_idx = 0;
    uint32_t level_start_idx = 2;

    while (n_level_leaves > (1 << cap_height)) {
        for (uint32_t i=0; i<n_level_leaves; i++) {
            uint32_t left = last_level_start_idx + i*(1<<(level+1));
            uint32_t right = left + 1;
            uint32_t to = (level_start_idx + (i>>1)*(1<<(level+2))) | (i&0b1);
            two_to_one(digests_cap, left, right, to);
        }

        level += 1;
        n_level_leaves = n_level_leaves >> 1;
        last_level_start_idx = level_start_idx;
        level_start_idx += (1<<level);
    }

    // caps
    uint32_t left = last_level_start_idx;
    uint32_t right = left + 1;
    uint32_t to = level_start_idx;
    two_to_one(digests_cap, left, right, to);

    return;
}

void print_leaves(F* leaves, uint32_t n_leaves) {
    for (int i=0; i<n_leaves; i++) {
        std::cout << std::dec;
        std::cout << "leave" << i << " is [";
        std::cout << std::hex;
        for (int j=0; j<LEAVE_WIDTH; j++) {
            std::cout << leaves[i*LEAVE_WIDTH + j] << ", ";
        }
        std::cout << "]" << std::endl;
    }
    std::cout << std::endl;
    std::cout << std::dec;
}

void print_digests(F* digests, uint32_t n_digests) {
    for (int i=0; i<n_digests; i++) {
        std::cout << std::dec;
        std::cout << "digest" << i << " is [";
        std::cout << std::hex;
        for (int j=0; j<HASH_WIDTH; j++) {
            std::cout << digests[i*HASH_WIDTH + j] << ", ";
        }
        std::cout << "]" << std::endl;
    }
    std::cout << std::endl;
    std::cout << std::dec;
}

void print_cap(F* digests_cap, uint32_t n_digests, uint32_t cap_height) {
    std::cout << std::hex;
    for (int i=0; i<(1<<cap_height); i++) {
        for (int j=0; j<HASH_WIDTH; j++) {
            std::cout << digests_cap[(n_digests+i)*HASH_WIDTH + j] << ", ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
    std::cout << std::dec;
}
